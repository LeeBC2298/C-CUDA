#include "hip/hip_runtime.h"
#include "kernel.h"
#include <iostream>

using namespace std;

__global__ void kernel(unsigned char* _src_dev, float* _dst_dev, int _src_width, int _src_height, int _dst_width, int _dst_height, float mean, float std){
    double srcXf;
    double srcYf;
    int srcX;
    int srcY;
    int dst_offset;
    int src_offset;

    int y = blockIdx.y*blockDim.y+threadIdx.y;
    int x = blockIdx.x*blockDim.x+threadIdx.x;
    if(x<_dst_width&&y<_dst_height){
        srcXf = x*((float)_src_width/_dst_width);
        srcYf = y*((float)_src_height/_dst_height);
        srcX = (int)srcXf;
        srcY = (int)srcYf;
        dst_offset = (y*_dst_width+x)*3;
        src_offset = (srcY*_src_width+srcX)*3;

        _dst_dev[dst_offset+0] = float((_src_dev[src_offset+2]-mean)/std);
        _dst_dev[dst_offset+1] = float((_src_dev[src_offset+1]-mean)/std);
        _dst_dev[dst_offset+2] = float((_src_dev[src_offset+0]-mean)/std);
    }
}

void preprocess_gpu(unsigned char* src, int ORIGN_W, int ORIGN_H, float* dst, int TARGET_W, int TARGET_H, float mean, float std){
    //hipSetDevice(9);
    unsigned char *src_dev;
    float *dst_dev;
    hipMalloc((void**)&src_dev, 3*ORIGN_W*ORIGN_H*sizeof(unsigned char));
    hipMalloc((void**)&dst_dev, 3*TARGET_W*TARGET_H*sizeof(float));

    /*float time_copy = 0;
    hipEvent_t start_copy, end_copy;
    hipEventCreate(&start_copy);
    hipEventCreate(&end_copy);

    hipEventRecord(start_copy, 0);*/
    hipMemcpy(src_dev, src, 3*ORIGN_W*ORIGN_H*sizeof(unsigned char), hipMemcpyHostToDevice);
    /*hipEventRecord(end_copy, 0);
    hipEventSynchronize(start_copy);
    hipEventSynchronize(end_copy);
    hipEventElapsedTime(&time_copy, start_copy, end_copy);
    hipEventDestroy(start_copy);
    hipEventDestroy(end_copy);
    cout << "gpu copy time: " << time_copy << endl;*/

    //dim3 grid(TARGET_H, TARGET_W);
    //kernel<<<grid,1>>>(src_dev, dst_dev, ORIGN_H, ORIGN_W, TARGET_H, TARGET_W);
    int unit = 16;
    dim3 grid((TARGET_W+unit-1)/unit,(TARGET_H+unit-1)/unit, 1);
    dim3 block(unit, unit, 1);

    /*float time_elapsed = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);*/
    kernel<<<grid,block>>>(src_dev, dst_dev, ORIGN_W, ORIGN_H, TARGET_W, TARGET_H, mean, std);
    hipDeviceSynchronize();
    /*hipEventRecord(stop, 0);

    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_elapsed, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop)
    cout << "gpu cal time: " << time_elapsed << endl;;*/

    hipMemcpy(dst, dst_dev, 3*TARGET_W*TARGET_H*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(src_dev);
    hipFree(dst_dev);
}
